#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

__global__
void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
    int N = 20 * (1 << 20);
    float* x, * y, * d_x, * d_y;
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start);

    // Perform SAXPY on 1M elements
    saxpy << <(N + 511) / 512, 512 >> > (N, 2.0f, d_x, d_y);

    hipEventRecord(stop);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 4.0f));
    }

    printf("Max error: %f\n", maxError);
    // N*4 is the number of bytes trasferred per array read or write
    // and the factor of three represents the readif of x and the reading of writing of y
    printf("Effective Bandwidth (GB/s): %f\n", N * 4 * 3 / milliseconds / 1e6); 
    

    free(x);
    free(y);
    hipFree(d_x);
    hipFree(d_y);
    
    return 0;
}