#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>

__global__ void saxpy(int n, float a, float* x, float* y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) y[i] = a * x[i] + y[i];
}
int main() {
	int N = 20 * (1 << 20);
	float* x, * y, * d_x, * d_y;
	x = (float*)malloc(N * sizeof(float));
	y = (float*)malloc(N * sizeof(float));
	hipMalloc(&d_x, N * sizeof(float));
	hipMalloc(&d_y, N * sizeof(float));

	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
	
	hipEventRecord(start);
	saxpy << <(N + 255) / 256, 256 >> > (N, 2.0, d_x, d_y);
	hipEventRecord(stop);

	hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);
	printf("Calculation time: %fms\n", miliseconds);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) {
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	}
	if (errAsync != hipSuccess) {
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
		//printf("Async kernel error: %s\n", hipGetErrorString(hipGetLastError()))
	}



	int nDevices;

	hipError_t err = hipGetDeviceCount(&nDevices);

	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
	for (int i = 0; i < nDevices; ++i) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("\tDevice name: %s\n", prop.name);
		printf("\tMemory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("\tMemory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("\tPeak Memory Bandwidth (GB/s): %f\n\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}

	free(x);
	free(y);
	hipFree(d_x);
	hipFree(d_y);

	return 0;
}