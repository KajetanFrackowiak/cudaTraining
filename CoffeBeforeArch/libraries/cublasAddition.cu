﻿
#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>

// Initialize a vector
void vector_init(float* a, int n) {
	for (int i = 0; i < n; ++i) {
		a[i] = (float)(rand() % 100);
	}
}

void verify_results(float* a, float* b, float* c, float factor, int n) {
	for (int i = 0; i < n; ++i) {
		assert(c[i] == factor * a[i] + b[i]);
	}

}

int main() {
	int n = 1 << 16;
	size_t bytes = n * sizeof(float);

	float* h_a, * h_b, * h_c;
	float* d_a, * d_b;

	h_a = (float*)malloc(bytes);
	h_b = (float*)malloc(bytes);
	h_c = (float*)malloc(bytes);
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);

	vector_init(h_a, n);
	vector_init(h_b, n);

	// Create and initialize a new context
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Copy the vectors over the device
	hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
	hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

	// Launch simple saxpy kernel (single precision a * x + y
	const float scale = 2.0f;
	hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1);

	// Copy the result vector back out
	hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

	verify_results(h_a, h_b, h_c, scale, n);

	// Clean up the created handle
	hipblasDestroy(handle);

	hipFree(d_a);
	hipFree(d_b);
	free(h_a);
	free(h_b);
	free(h_c);

	printf("COMPLETED SUCCESSFULLY\n");

	return 0;
}
