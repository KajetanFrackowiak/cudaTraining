﻿#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <math.h>

void verify_solution(float* a, float* b, float* c, int n) {
	float temp;
	float epsilon = 1e-3;
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			temp = 0;
			for (int k = 0; k < n; ++k) {
				temp += a[k * n + i] * b[j * n + k];
			}
			assert(fabs(c[j * n + i] - temp) < epsilon);
		}
	}
}

int main() {
	int n = 1 << 10;
	size_t bytes = n * n * sizeof(float);

	float* h_a, * h_b, * h_c;
	float* d_a, * d_b, * d_c;

	h_a = (float*)malloc(bytes);
	h_b = (float*)malloc(bytes);
	h_c = (float*)malloc(bytes);
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Pseudo random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

	// Fill the matrix with random numbers on the device
	hiprandGenerateUniform(prng, d_a, n * n);

	// cuBLAS handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Scaling factors;
	float alpha = 1.0f;
	float beta = 0.0f;

	// Calcuale: c = (alpha * a) * b + (beta*c)
	// (m X n) * (n X k) = (m X k)
	// Signature: handle, operation, m, n, k, alpha, A, Ida, B ldb, beta, C ldc
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);

	hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	verify_solution(h_a, h_b, h_c, n);

	printf("COMPLETED SUCCESSFULLY!\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}