#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    // Calculate global thead ID (tid)
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Vector boundary guard
    if (tid < n) {
        // Each thread adds a single element
        c[tid] = a[tid] + b[tid];
    }
}

void matrix_init(int* matrix, int n) {
    for (int i = 0; i < n; i++) {
        matrix[i] = rand() % 100; // random value between 0 and 99
    }
}

void error_check(int* a, int* b, int* c, int n) {
    for (int i = 0; i < n; ++i) {
        assert(c[i] == a[i] + b[i]);
    }
}

int main() {
    // Vector size of 2^16 (65536 elements)
    int n = 2 << 16;
    // Host vector pointers
    int* h_a, * h_b, * h_c;
    // Device vector pointer
    int* d_a, * d_b, * d_c;
    // Allocation size for all vectors
    size_t bytes = sizeof(int) * n;

    // Allocate host memory
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // Allocate device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Initialize vectors a and b with random values between 0 and 99
    matrix_init(h_a, n);
    matrix_init(h_b, n);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Threadblock size
    int NUM_THREADS = 256;

    // Grid size
    int NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;

    // Launch kernel on default stream w/o shmem
    vectorAdd << <NUM_BLOCKS, NUM_THREADS >> > (d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Check result for errors
    error_check(h_a, h_b, h_c, n);

    printf("COMPLETED SUCCESSFULLY\n");

    // Free host and device memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
