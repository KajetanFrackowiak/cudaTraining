#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>

// Check result
void verify_result(int* a, int* b, int* c, int n) {
	int* verify_c;
	verify_c = (int*)malloc(n * n * sizeof(int));
	for (int i = 0; i < n; i++) { // all the rows
		for (int j = 0; j < n; j++) { // all the columns
			for (int k = 0; k < n; k++) { // every single elem in the rows and columns
				verify_c[i * n + j] += a[i * n + k] * b[k * n + j];
			}
		}
	}

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			assert(c[i * n + j] == verify_c[i * n + j]);
		}
	}
}

int main() {
	// Matrix size of 1024 x 1024
	int n  = 1 << 10;

	// Size (in bytes) of matrix
	size_t bytes = n * n * sizeof(int);

	// Host pointers
	int* h_a, * h_b, * h_c;

	// Allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	// Device pointers
	int* d_a, *d_b, *d_c;

	// Allocate device memory	
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Initialize matrices
	init_matrices(h_a, h_b, n);

	// Copy data to the device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// Threads per block
	int BLOCK_SIZE = 16;

	// Blocks in each dimension
	int GRID_SIZE = (int)ceil(n / BLOCK_SIZE);

	// Use dim3 objects
	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel
	matrixMul << <grid, threads >> > (d_a, d_b, d_c, n);

	// Copy back to the host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
	
	// Check results
	verify_result(h_a, h_b, h_c, n);

	printf("COMPLETED SUCESSFULLY\n");

	return 0;
}
