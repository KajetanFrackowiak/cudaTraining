#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

inline
hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined (_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

__global__ void vecAdd(float* A, float* B, float* C, int numElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements) {
		C[i] = A[i] + B[i];
	}
}

int main(void) {
	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);

	float* h_A, * h_B, * h_C;
	h_A = (float*)malloc(size);
	h_B = (float*)malloc(size);
	h_C = (float*)malloc(size);


	for (int i = 0; i < numElements; ++i) {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	float* d_A, * d_B, * d_C;
	checkCuda(hipMalloc((void**)&d_A, size));
	checkCuda(hipMalloc((void**)&d_B, size));
	checkCuda(hipMalloc((void**)&d_C, size));

	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	float ms; // elapsed time in miliseconds
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	checkCuda(hipEventRecord(startEvent, 0));
	checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

	vecAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);

	checkCuda(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	printf("Time for sequential transfer and execute (ms): %f\n", ms);

	checkCuda(hipFree(d_A));
	checkCuda(hipFree(d_B));
	checkCuda(hipFree(d_C));
	free(h_A);
	free(h_B);
	free(h_C);
	
	return 0;
}